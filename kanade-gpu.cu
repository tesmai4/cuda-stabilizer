#include "hip/hip_runtime.h"
#ifndef KANADE_NO_GPU

namespace kgpu {

const unsigned BLOCK_DIM = 32;

#include "kanade.h"
#include "hip/hip_runtime_api.h"

unsigned char* ioFrame24 = NULL;									// pamiec pod ramke 24b - uzywana na wejsciu i wyjsciu
unsigned char* ioFrame32 = NULL;									// pamiec pod ramke 32b - uzywana na wejsciu jako tymczasowa dla tekstury
unsigned char* ioFrame8[PYRAMID_SIZE] = { NULL };					// pamiec pod piramide ramke 8b  - uzywana na wejsciu jako tymczasowa dla tekstury
unsigned char* prevFrame8[PYRAMID_SIZE] = { NULL };					// poprzednia piramida ramek w formacie osmiobitowym - do wyliczania dt

unsigned char* gpuFrame;											// ramka oryginalna, ktora bedzie uzywana do przesuwania

float* devDx = NULL;												// pamiec na skladowe dx, dy i dt
float* devDy = NULL;
float* devDt = NULL;

float* cpuDx = NULL;
float* cpuDy = NULL;
float* cpuDt = NULL;

float* devG = NULL;
float* devB = NULL;
float* cpuG = NULL;
float* cpuB = NULL;

texture<uchar4, 2, hipReadModeElementType> frameTex;				// kolorowa ramka wej�ciowa do przesuwania	
hipArray* frameTexMem = NULL;										

template <typename T>
__device__ inline unsigned char toColor(T value)
{
	if (value > 255) return 255;
	if (value < 0) return 0;
	return (unsigned char) value;
}

template <>
__device__ inline unsigned char toColor<unsigned>(unsigned value)
{
	if (value > 255) return 255;
	return (unsigned char) value;
}

__global__ void prepareFrame(unsigned char* frame24, unsigned char* frame32, unsigned char* frame8, unsigned width, unsigned height)
{
	// @todo: cala te funkcje mozna zrobic wydajniej przy uzyciu pamieci dzielonej

	int pos = blockIdx.x*blockDim.x + threadIdx.x;
	if (pos >= width*height) return;

	unsigned char r = frame24[3*pos];
	unsigned char g = frame24[3*pos+1];
	unsigned char b = frame24[3*pos+2];

	// konwersja na skale szarosci wg. jasnosci piksela
	//frame8[pos] = toColor(floor(0.299f * r + 0.587 * g + 0.114 * b));		// ten sposob (przeksztalcenie na YUV) powodowal bledy miedzy implementacja CPU a GPU
	//frame8[pos] = toColor(floor((r + g + b)/3.0f));
	frame8[pos] = MAX(MAX(r, g), b);
	
	// konwersja formatu 24b na 32b dla kompatybilnosci z teksturami (wartosc alfa jest zerowana przed wywolaniem)
	if (frame32 != NULL)
	{
		frame32[4*pos] = r;
		frame32[4*pos+1] = g;
		frame32[4*pos+2] = b;
	}
}

__global__ void build_pyramid_level(unsigned char* prevLvl8b, unsigned char* newLvl8b, unsigned newWidth, unsigned newHeight)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	// zezwol tylko na wartosci z poprawnego zakresu
	if (x >= newWidth || y >= newHeight) return;

	// srednia z czterech wartosci, mozna tez zrobic min/max
	// @todo: to mozna zrobic wydajniej laczac operacje odczytu

	int width = 2*newWidth;
	int np = 2*(y*width + x);
	newLvl8b[y*newWidth+x] = toColor(
		((unsigned)prevLvl8b[np] + (unsigned)prevLvl8b[np+1] + 
		 (unsigned)prevLvl8b[np+width] + (unsigned)prevLvl8b[np+width+1]) / 4);
}

__global__ void translate(float vx, float vy, unsigned char* input24, unsigned char* output24, unsigned width, unsigned height)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	
	// zezwol tylko na wartosci z poprawnego zakresu
	if (x >= width || y >= height) return;

	// werja nearest neighbour
	int dx = (int) round(vx);					
	int dy = (int) round(vy);

	unsigned dpos = 3*(y * width + x);

	if (x-dx >= 0 && y-dy >= 0 && x-dx < width-1 && y-dy < height-1)
	{
		unsigned spos = 3*((y - dy) * width + x - dx);
		for (unsigned c=0; c<3; c++) 
		{
			output24[dpos+c] = input24[spos+c];
		}
	}

	// wersja interpolujaca
/*
	int dx = (int) floor(vx);					// @todo: te obliczenia mozna sprobowac wykonac na cpu i wrzucic do pamieci stalej
	int dy = (int) floor(vy);

	double nx = vx - floor(vx);					// nx - fraction of the next pixel in x taken into interpolation
	double ny = vy - floor(vy);
	double tx = 1 - nx;							// tx - fraction of thix pixel taken into interpolation
	double ty = 1 - ny;

	unsigned dpos = 3*(y * width + x);

	if (x-dx >= 0 && y-dy >= 0 && x-dx < width-1 && y-dy < height-1)
	{
		unsigned spos = 3*((y - dy) * width + x - dx);
		for (unsigned c=0; c<3; c++) 
		{
			float y1 = (tx * input24[spos+c] + nx * input24[spos+c+3]);
			float y2 = (tx * input24[spos+c+3*width] + nx * input24[spos+c+3*width+3]);

			output24[dpos+c] = (unsigned char)(ty * y1 + ny * y2);
		}
	}
*/
	//uchar4 c = tex2D(frameTex, x - vx, y - vy);

	// @todo: to mozna zrobic wydajniej przy uzyciu pamieci dzielonej
	//int pos = 3*(y*width + x);
	//output24[pos] = c.x;
	//output24[pos+1] =  c.y;
	//output24[pos+2] =  c.z;
}

__global__ void calculate_dxdy(unsigned char* frame8, float* dx, float* dy, unsigned width, unsigned height)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	
	// zezwol tylko na wartosci z poprawnego zakresu
	if (x >= width || y >= height) return;

	// rozwazyc czy nie zastapic tego dodaniem paddingu po obu stronach obrazu

	// @todo: to mozna zrobic wydajniej przy uzyciu pamieci dzielonej
	int pos = y*width + x;
	if (x != 0 && x != width-1)
		dx[pos] = ((float)(frame8[pos+1]-frame8[pos-1])) / 2.0f;
	else
		dx[pos] = 0;

	if (y != 0 && y != height-1)
		dy[pos] = ((float)(frame8[pos+width]-frame8[pos-width])) / 2.0f;
	else
		dy[pos] = 0;	
}

__global__ void calculate_dt(unsigned pyrLvl, float vx, float vy, unsigned char* prevFrame8b, unsigned char* currFrame8b, float* dt, unsigned width, unsigned height)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	
	// zezwol tylko na wartosci z poprawnego zakresu
	if (x >= width || y >= height) return;

	int dx = (int) floor(vx);
	int dy = (int) floor(vy);

	double nx = vx - floor(vx);					// fractions of the next pixel taken into interpolation
	double ny = vy - floor(vy);
	double tx = 1 - nx;							// fraction of this pixel taken into interpolation
	double ty = 1 - ny;
	
	float interpolated = 0;
	if (x+dx >= 0 && y+dy >= 0 && x+dx < width-1 && y+dy < height-1)
	{
		unsigned spos = (y + dy) * width + x + dx;
		interpolated = (unsigned char)((tx * currFrame8b[spos] + nx * currFrame8b[spos + 1] + ty * currFrame8b[spos] + ny * currFrame8b[spos + width]) / 2.0f);		// na 2 a nie na 4, bo korzystamy z "polkolorow" (w punkcie srodkowym kazdy kolor jest mnozony przez 0.5)
	}

	int dpos = y*width + x;
	dt[dpos] = (float)prevFrame8b[dpos] - interpolated;
}

// for full image operations
template <typename T1, typename T2>
__device__ __host__ float dotSum(T1* a, T2* b, int size)
{
	float sum = 0.0f;
	for (int i=0; i<size; i++)
		sum += a[i] * b[i];
	return sum;
}

// for window operations
template <typename T1, typename T2>
__device__ __host__ float dotSum(T1* a, T2* b, int width, int height)
{
	float sum = 0.0f;
	int px = width/2;
	int py = height/2;

	int sx = MAX(0, px-(int)WINDOW_SIZE_DIV_2);
	int ex = MIN(width-1, px+WINDOW_SIZE_DIV_2);
	int sy = MAX(0, py-(int)WINDOW_SIZE_DIV_2);
	int ey = MIN(height-1, py+WINDOW_SIZE_DIV_2);

	for (int y=sy; y<=ey; y++)
		for (int x=sx; x<=ex; x++)
		{
			int pos = y*width+x;
			sum += a[pos] * b[pos];
		}
	
	return sum;
}

// reduce to g

/*
__device__ unsigned int kernel_block_counter = 0;
__shared__ bool kernel_last_block_done;

__device__ float g_partial_sum(float* dx, float* dy)
{
	__shared__ float t_dxdx[BLOCK_DIM];
	__shared__ float t_dxdy[BLOCK_DIM];
	__shared__ float t_dydy[BLOCK_DIM];

	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	float l_dx = dx[idx];
	float l_dy = dy[idx];

	t_dxdx[threadIdx.x] = l_dx * l_dx;
	t_dxdy[threadIdx.x] = l_dx * l_dy;
	t_dydy[threadIdx.x] = l_dy * l_dy;

	__syncthreads();

	int nTotalThreads = blockDim.x;					 
	while (nTotalThreads > 1)
	{
		int halfPoint = (nTotalThreads >> 1);	// divide by two		
 
		if (threadIdx.x < halfPoint)
		    t_dxdx[threadIdx.x] += t_dxdx[threadIdx.x + halfPoint];
 
		__syncthreads();
 
		nTotalThreads = halfPoint;
	}

	return t_dxdx[0];
}

__device__ float g_total_sum(float* dxdx, float* dxdy, float* dydy)
{
	float sum = 0;
	for (int i=0; i<gridDim.x; i++)
		sum += dxdx[i];
	return sum;
}

__global__ void reduce_to_g(float* dx, float* dy, float* dxdx, float* dxdy, float* dydy, unsigned size)
{
	float partialSum = g_partial_sum(dx, dy);
	if (threadIdx.x == 0)
	{
		dxdx[blockIdx.x] = partialSum;
		__threadfence();
		unsigned int value = atomicInc(&kernel_block_counter, gridDim.x);
		kernel_last_block_done = (value == (gridDim.x - 1));
	}

	__syncthreads();

	if (kernel_last_block_done) 
	{
		float totalSum = g_total_sum(dxdx, dxdy, dydy);
		if (threadIdx.x == 0) 
		{
			dxdx[0] = totalSum;
			kernel_block_counter = 0;
		}
	}

	
}
*/

// ta wersja wprowadzala znaczaco duze bledy
__global__ void reduce_to_g(float* dx, float* dy, float* dxdx, float* dxdy, float* dydy, unsigned size)
{
	// to tez nie dziala
	/*int idx = threadIdx.x + blockIdx.x * blockDim.x;

	float s1 = dx[idx]*dx[idx];
	float s2 = dx[idx]*dy[idx];
	float s3 = dy[idx]*dy[idx];
	atomicAdd(dxdx,s1);
	atomicAdd(dxdy,s2);
	atomicAdd(dydy,s3);
	*/
	
	__shared__ float t_dxdx[BLOCK_DIM];
	__shared__ float t_dxdy[BLOCK_DIM];
	__shared__ float t_dydy[BLOCK_DIM];

	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	float l_dx = dx[idx];
	float l_dy = dy[idx];

	t_dxdx[threadIdx.x] = l_dx * l_dx;
	t_dxdy[threadIdx.x] = l_dx * l_dy;
	t_dydy[threadIdx.x] = l_dy * l_dy;

	__syncthreads();

	if (threadIdx.x == 0) 
	{
		float s_dxdx = 0, s_dxdy = 0, s_dydy = 0;

		#pragma unroll
		for (int i = 0; i < BLOCK_DIM; i++)
		{
			s_dxdx += t_dxdx[i];
			s_dxdy += t_dxdy[i];
			s_dydy += t_dydy[i];
		}

		//printf("%f\n", s_dxdx);

		atomicAdd(dxdx, s_dxdx);
		atomicAdd(dxdy, s_dxdy);
		atomicAdd(dydy, s_dydy);
	}
	
}

// na chwile obecna nie wykorzystywane, bo strasznie mulilo - jesli to robic to jakos zmyslniej
__global__ void reduce_to_b(float* dx, float* dy, float* dt, float* bx, float* by)
{
	__shared__ float t_dxdt[BLOCK_DIM];
	__shared__ float t_dydt[BLOCK_DIM];

	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	float l_dt = dt[idx];

	t_dxdt[threadIdx.x] = dx[idx] * l_dt;
	t_dydt[threadIdx.x] = dy[idx] * l_dt;

	__syncthreads();

	if (threadIdx.x == 0) 
	{
		float s_dxdt = 0, s_dydt = 0;

		#pragma unroll
		for (int i = 0; i < BLOCK_DIM; i++)
		{
			s_dxdt += t_dxdt[i];
			s_dydt += t_dydt[i];
		}

		atomicAdd(bx, s_dxdt);
		atomicAdd(by, s_dydt);
	}
}

/*
 * w wiekszosci przypadkow pamiec alokowana jest tylko raz na caly film, ale np. ioFrame8 jest alokowane co klatke
 * (po to, zeby nie kopiowac zawartosci poprzedniej ramki, przypisujemy jej pointer, a pod nowa ramke alokujemy
 * nowa pamiec)
 */

void allocateMemoryIfNeeded(unsigned width, unsigned height)
{
	if (ioFrame24 == NULL)
		checkCudaErrors(hipMalloc(&ioFrame24, 3 * width * height * sizeof(unsigned char))); 

	for (unsigned i=0; i<PYRAMID_SIZE; ++i)
	{
		if (ioFrame8[i] == NULL)
			checkCudaErrors(hipMalloc(&ioFrame8[i], width * height * sizeof(unsigned char))); 
	}
		
	if (ioFrame32 == NULL)
		checkCudaErrors(hipMalloc(&ioFrame32, 4 * width * height * sizeof(unsigned char))); 

	if (gpuFrame == NULL)
		checkCudaErrors(hipMalloc(&gpuFrame, 3 * width * height * sizeof(unsigned char))); 

	unsigned wthAligned = width * height;
	if (wthAligned % BLOCK_DIM != 0)
		wthAligned += (BLOCK_DIM - wthAligned % BLOCK_DIM);

	if (devDx == NULL)
		checkCudaErrors(hipMalloc(&devDx, wthAligned * sizeof(float))); 

	if (devDy == NULL)
		checkCudaErrors(hipMalloc(&devDy, wthAligned * sizeof(float))); 

	if (devDt == NULL)
		checkCudaErrors(hipMalloc(&devDt, wthAligned * sizeof(float))); 

	if (cpuDx == NULL)
		cpuDx = (float*) malloc(width * height * sizeof(float));

	if (cpuDy == NULL)
		cpuDy = (float*) malloc(width * height * sizeof(float));

	if (cpuDt == NULL)
		cpuDt = (float*) malloc(width * height * sizeof(float));	
}

void kanadeNextFrame(unsigned char* pixels, unsigned width, unsigned height)
{
	for (unsigned i=0; i<PYRAMID_SIZE; ++i)
	{
		if (ioFrame8[i] != NULL)
		{			
			if (prevFrame8[i] == NULL)
			{
				prevFrame8[i] = ioFrame8[i];
				ioFrame8[i] = NULL;
			}
		}
	}

	allocateMemoryIfNeeded(width, height);

	if (frameTexMem != NULL) 
	{
		checkCudaErrors(hipUnbindTexture(frameTex));
		hipFreeArray(frameTexMem);
	}

	// przygotuj dane wejsciowe pod tekstury
	checkCudaErrors(hipMemcpy(ioFrame24, pixels, 3 * width * height * sizeof(unsigned char), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemset(ioFrame32, 0, 4 * width * height * sizeof(unsigned char)));
	prepareFrame<<<(width*height + BLOCK_DIM-1)/BLOCK_DIM, BLOCK_DIM>>>(ioFrame24, ioFrame32, ioFrame8[0], width, height);
	checkCudaErrors(hipGetLastError());

	checkCudaErrors(hipMemcpy(gpuFrame, ioFrame24, 3 * width * height * sizeof(unsigned char), hipMemcpyDeviceToDevice));

	// przygotuj tekstury
	/*hipChannelFormatDesc desc24b = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned); 	 
	checkCudaErrors(hipMallocArray(&frameTexMem, &desc24b, width, height)); 
	checkCudaErrors(hipDeviceSynchronize());	
	checkCudaErrors(hipMemcpyToArray(frameTexMem, 0, 0, ioFrame32, 4 * width * height * sizeof(unsigned char), hipMemcpyDeviceToDevice)); 
	checkCudaErrors(hipBindTextureToArray(frameTex, frameTexMem, desc24b)); */
	
}

void kanadePrepareForNextFrame(unsigned width[PYRAMID_SIZE], unsigned height[PYRAMID_SIZE])
{
	/*
	 * right now we have the translated frame in ioFrame24 (as this function is called at the end of kanadeExecute())
	 * Consequently, we will use the translated frame as a reference frame for the next frame. Hence, we have to 
	 * build the pyramid (again) for the translated frame, duh. 
	 */

	// prepare pyramid level zero
	prepareFrame<<<(width[0]*height[0] + BLOCK_DIM-1)/BLOCK_DIM, BLOCK_DIM>>>(ioFrame24, NULL, prevFrame8[0], width[0], height[0]);
	checkCudaErrors(hipDeviceSynchronize());

	// prepare other pyramid levels
	for (unsigned i=1; i<PYRAMID_SIZE; i++)
	{
		dim3 dimGrid((width[i] + BLOCK_DIM - 1) / BLOCK_DIM, (height[i] + BLOCK_DIM - 1) / BLOCK_DIM);	
		dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);	

		build_pyramid_level<<<dimGrid, dimBlock>>>(prevFrame8[i-1], prevFrame8[i], width[i], height[i]);

		// musimy sie synchronizowac, bo kolejne poziomy wymagaja obliczen z popzednich
		checkCudaErrors(hipDeviceSynchronize());
	}
}

void kanadeTranslate(unsigned char* target, float vx, float vy, unsigned width, unsigned height)
{	
	dim3 dimGrid((width + BLOCK_DIM - 1) / BLOCK_DIM, (height + BLOCK_DIM - 1) / BLOCK_DIM);
	dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);	

	hipMemset(ioFrame24, 0, 3*width*height*sizeof(unsigned char));
	translate<<<dimGrid, dimBlock>>>(vx, vy, gpuFrame, ioFrame24, width, height);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipMemcpy(target, ioFrame24, 3 * width * height * sizeof(unsigned char), hipMemcpyDeviceToHost));
}

void kanadeCalculateG(unsigned pyrLvl, unsigned width, unsigned height, float& dxdx, float& dxdy, float& dydy)
{
	// potrzebujemy wyrownac do BLOCK_DIM, zeby jadro redukcji dzialalo poprawnie
	unsigned sizeAligned = width * height;
	if (sizeAligned % BLOCK_DIM != 0)
		sizeAligned += (BLOCK_DIM - sizeAligned % BLOCK_DIM);
	checkCudaErrors(hipMemset(devDx, 0, sizeAligned * sizeof(float)));
	checkCudaErrors(hipMemset(devDy, 0, sizeAligned * sizeof(float)));

	dim3 dimGrid((width + BLOCK_DIM - 1) / BLOCK_DIM, (height + BLOCK_DIM - 1) / BLOCK_DIM);
	dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);	
	calculate_dxdy<<<dimGrid, dimBlock>>>(prevFrame8[pyrLvl], devDx, devDy, width, height);
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipMemset(devG, 0, 3 * sizeof(float)));
	reduce_to_g<<<sizeAligned / BLOCK_DIM, BLOCK_DIM>>>(devDx, devDy, &devG[0], &devG[1], &devG[2], sizeAligned);
	checkCudaErrors(hipGetLastError());

	//reduce_to_g<<<(width * height + REDUCTION_BLOCK_DIM - 1) / REDUCTION_BLOCK_DIM, REDUCTION_BLOCK_DIM>>>(devDx, devDy, &devG[0], NULL, NULL, sizeAligned);

	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipMemcpy(cpuG, devG, 3 * sizeof(float), hipMemcpyDeviceToHost));
	dxdx = cpuG[0];
	dxdy = cpuG[1];
	dydy = cpuG[2];

	
	/*
	// to poki co musi byc kopiowane dla obliczania wektora B
	unsigned size = width * height;
	checkCudaErrors(hipMemcpy(cpuDx, devDx, size * sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(cpuDy, devDy, size * sizeof(float), hipMemcpyDeviceToHost));

	dxdx = dotSum(cpuDx, cpuDx, size);
	dxdy = dotSum(cpuDx, cpuDy, size);
	dydy = dotSum(cpuDy, cpuDy, size);
	//dxdx = dotSum(cpuDx, cpuDx, width, height);
	//dxdy = dotSum(cpuDx, cpuDy, width, height);
	//dydy = dotSum(cpuDy, cpuDy, width, height);
	*/
}

void kanadeCalculateB(unsigned pyrLvl, float vx, float vy, unsigned width, unsigned height, float& bx, float& by)
{
	unsigned sizeAligned = width * height;
	if (sizeAligned % BLOCK_DIM != 0)
		sizeAligned += (BLOCK_DIM - sizeAligned % BLOCK_DIM);
	checkCudaErrors(hipMemset(devDt, 0, sizeAligned * sizeof(float)));

	dim3 dimGrid((width + BLOCK_DIM - 1) / BLOCK_DIM, (height + BLOCK_DIM - 1) / BLOCK_DIM);
	dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);	
	calculate_dt<<<dimGrid, dimBlock>>>(pyrLvl, vx, vy, prevFrame8[pyrLvl], ioFrame8[pyrLvl], devDt, width, height);
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipMemset(devB, 0, 2 * sizeof(float)));
	reduce_to_b<<<sizeAligned / BLOCK_DIM, BLOCK_DIM>>>(devDx, devDy, devDt, &devB[0], &devB[1]);

	checkCudaErrors(hipMemcpy(cpuB, devB, 2 * sizeof(float), hipMemcpyDeviceToHost));
	bx = cpuB[0];
	by = cpuB[1];

	/*
	unsigned size = width * height;
	checkCudaErrors(hipMemcpy(cpuDt, devDt, size * sizeof(float), hipMemcpyDeviceToHost));

	bx = dotSum(cpuDx, cpuDt, size);
	by = dotSum(cpuDy, cpuDt, size);
	//bx = dotSum(cpuDx, cpuDt, width, height);
	//by = dotSum(cpuDy, cpuDt, width, height);
	*/
}

// level 0 is automatically initiated elsewhere
void kanadeBuildPyramidLevel(unsigned levelId, unsigned newWidth, unsigned newHeight)
{
	dim3 dimGrid((newWidth + BLOCK_DIM - 1) / BLOCK_DIM, (newHeight + BLOCK_DIM - 1) / BLOCK_DIM);	
	dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);	

	build_pyramid_level<<<dimGrid, dimBlock>>>(ioFrame8[levelId-1], ioFrame8[levelId], newWidth, newHeight);

	// musimy sie synchronizowac, bo kolejne poziomy wymagaja obliczen z popzednich
	checkCudaErrors(hipDeviceSynchronize());
}

void kanadeInit()
{
	hipDeviceProp_t deviceProp;
	int devID = 0;
	checkCudaErrors(hipSetDevice(devID));
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);	

	// jesli tekstura odwoluje sie poza swoj zakres to zwracany jest kolor zerowy
	frameTex.addressMode[0] = hipAddressModeBorder;	
    frameTex.addressMode[1] = hipAddressModeBorder; 

	checkCudaErrors(hipMalloc(&devG, 3 * sizeof(float))); 
	checkCudaErrors(hipMalloc(&devB, 2 * sizeof(float))); 
	
	cpuG = (float*) malloc(3 * sizeof(float));
	cpuB = (float*) malloc(2 * sizeof(float));	
}

void kanadeCleanup()
{
	if (frameTexMem != NULL)
		hipFreeArray(frameTexMem);

	for (unsigned i=0; i<PYRAMID_SIZE; i++)
	{
		if (ioFrame8[i] != NULL)
			hipFree(ioFrame8[i]);
		if (prevFrame8[i] != NULL)
			hipFree(prevFrame8[i]);
	}

	if (ioFrame24 != NULL)
		hipFree(ioFrame24);

	if (ioFrame32 != NULL)
		hipFree(ioFrame32);

	if (gpuFrame != NULL)
		hipFree(gpuFrame);

	if (devDx != NULL)
		hipFree(devDx);

	if (devDy != NULL)
		hipFree(devDy);

	if (devDt != NULL)
		hipFree(devDt);

	if (cpuDx != NULL)
		free(cpuDx);
	if (cpuDy != NULL)
		free(cpuDy);
	if (cpuDt != NULL)
		free(cpuDt);

	if (devG != NULL)
		hipFree(devG);
	if (devB != NULL)
		hipFree(devB);
	if (cpuG != NULL)
		free(cpuG);
	if (cpuB != NULL)
		free(cpuB);
}

// for testing purposes only
void getIoFrame8(unsigned char* target, unsigned lvl, unsigned width, unsigned height) 
{
	checkCudaErrors(hipMemcpy(target, ioFrame8[lvl], width * height * sizeof(unsigned char), hipMemcpyDeviceToHost));
}

void getPrevFrame8(unsigned char* target, unsigned lvl, unsigned width, unsigned height) 
{
	checkCudaErrors(hipMemcpy(target, prevFrame8[lvl], width * height * sizeof(unsigned char), hipMemcpyDeviceToHost));
}

void getDevDx(float* target, unsigned width, unsigned height)
{
	checkCudaErrors(hipMemcpy(target, devDx, width * height * sizeof(float), hipMemcpyDeviceToHost));
}

void getDevDy(float* target, unsigned width, unsigned height)
{
	checkCudaErrors(hipMemcpy(target, devDy, width * height * sizeof(float), hipMemcpyDeviceToHost));
}

void getDevDt(float* target, unsigned width, unsigned height)
{
	checkCudaErrors(hipMemcpy(target, devDt, width * height * sizeof(float), hipMemcpyDeviceToHost));
}

}

#endif