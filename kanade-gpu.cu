#include "hip/hip_runtime.h"
#ifndef KANADE_NO_GPU

namespace kgpu {

const unsigned BLOCK_DIM = 32;

#include "kanade.h"
#include "hip/hip_runtime_api.h"

unsigned char* cpuMem = NULL;										// pamiec CPU w ktorej przechowujemy ramke wynikowa

unsigned char* ioFrame24 = NULL;									// pamiec pod ramke 24b - uzywana na wejsciu i wyjsciu
unsigned char* ioFrame32 = NULL;									// pamiec pod ramke 32b - uzywana na wejsciu jako tymczasowa dla tekstury
unsigned char* ioFrame8[PYRAMID_SIZE] = { NULL };					// pamiec pod piramide ramke 8b  - uzywana na wejsciu jako tymczasowa dla tekstury
unsigned char* prevFrame8[PYRAMID_SIZE] = { NULL };					// poprzednia piramida ramek w formacie osmiobitowym - do wyliczania dt

unsigned char* gframe;												// ramki w skali szarosci do algorytmu L-K
unsigned char* gref;

float* devDx = NULL;												// pamiec na skladowe dx, dy i dt
float* devDy = NULL;
float* devDt = NULL;

texture<uchar4, 2, hipReadModeElementType> frameTex;				// kolorowa ramka wej�ciowa do przesuwania	
hipArray* frameTexMem = NULL;										

texture<uchar1, 2, hipReadModeElementType> tex8b[PYRAMID_SIZE];
hipArray* tex8bMem[PYRAMID_SIZE] = { NULL };

template <typename T>
__device__ inline unsigned char toColor(T value)
{
	if (value > 255) return 255;
	if (value < 0) return 0;
	return (unsigned char) value;
}

__global__ void prepareFrame(unsigned char* frame24, unsigned char* frame32, unsigned char* frame8, unsigned width, unsigned height)
{
	// @todo: cala te funkcje mozna zrobic wydajniej przy uzyciu pamieci dzielonej

	int pos = blockIdx.x*blockDim.x + threadIdx.x;
	if (pos >= width*height) return;

	unsigned char r = frame24[3*pos];
	unsigned char g = frame24[3*pos+1];
	unsigned char b = frame24[3*pos+2];

	// konwersja na skale szarosci wg. jasnosci piksela
	frame8[pos] = toColor(0.299f * r + 0.587 * g + 0.114 * b);
	
	// konwersja formatu 24b na 32b dla kompatybilnosci z teksturami (wartosc alfa jest zerowana przed wywolaniem)
	frame32[4*pos] = r;
	frame32[4*pos+1] = g;
	frame32[4*pos+2] = b;
}

__global__ void build_pyramid_level(unsigned char* prevLvl8b, unsigned char* newLvl8b, unsigned newWidth, unsigned newHeight)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	// zezwol tylko na wartosci z poprawnego zakresu
	if (x >= newWidth || y >= newHeight) return;

	// srednia z czterech wartosci, mozna tez zrobic min/max
	// @todo: to mozna zrobic wydajniej laczac operacje odczytu

	int width = 2*newWidth;
	int np = 2*(y*width + x);
	newLvl8b[y*newWidth+x] = toColor(
		((unsigned)prevLvl8b[np] + (unsigned)prevLvl8b[np+1] + 
		 (unsigned)prevLvl8b[np+width] + (unsigned)prevLvl8b[np+width+1]) / 4);
}

__global__ void translate(float vx, float vy, unsigned char* frame24, unsigned width, unsigned height)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	
	// zezwol tylko na wartosci z poprawnego zakresu
	if (x >= width || y >= height) return;

	uchar4 c = tex2D(frameTex, x - vx, y - vy);

	// @todo: to mozna zrobic wydajniej przy uzyciu pamieci dzielonej
	int pos = 3*(y*width + x);
	frame24[pos] = c.x;
	frame24[pos+1] =  c.y;
	frame24[pos+2] =  c.z;
}

__global__ void calculate_dxdy(unsigned char* frame8, float* dx, float* dy, unsigned width, unsigned height)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	
	// zezwol tylko na wartosci z poprawnego zakresu
	if (x >= width || y >= height) return;

	// rozwazyc czy nie zastapic tego dodaniem paddingu po obu stronach obrazu

	// @todo: to mozna zrobic wydajniej przy uzyciu pamieci dzielonej
	int pos = y*width + x;
	if (x != 0 && x != width-1)
		dx[pos] = ((float)(frame8[pos+1]-frame8[pos-1])) / 2.0f;
	else
		dx[pos] = 0;

	if (y != 0 && y != height-1)
		dy[pos] = ((float)(frame8[pos+width]-frame8[pos-width])) / 2.0f;
	else
		dy[pos] = 0;	
}

__global__ void calculate_dt(unsigned pyrLvl, float vx, float vy, unsigned char* prevFrame8b, float* dt, unsigned width, unsigned height)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	
	// zezwol tylko na wartosci z poprawnego zakresu
	if (x >= width || y >= height) return;
	
	int pos = y*width + x;
	//dt[pos] = ((float)prevFrame8b[pos]) - ((float)tex2D(tex8b[pyrLvl], x + vx, y + vy));
	// @todo - trzeba explicite podac ptr tekstury tex8b
}

__global__ void reduce_to_g(float* dx, float* dy, float* dxdx, float* dxdy, float* dydy)
{
}

/*
 * w wiekszosci przypadkow pamiec alokowana jest tylko raz na caly film, ale np. ioFrame8 jest alokowane co klatke
 * (po to, zeby nie kopiowac zawartosci poprzedniej ramki, przypisujemy jej pointer, a pod nowa ramke alokujemy
 * nowa pamiec)
 */

void allocateMemoryIfNeeded(unsigned width, unsigned height)
{
	if (ioFrame24 == NULL)
		checkCudaErrors(hipMalloc(&ioFrame24, 3 * width * height * sizeof(unsigned char))); 

	for (unsigned i=0; i<PYRAMID_SIZE; ++i)
	{
		if (ioFrame8[i] == NULL)
			checkCudaErrors(hipMalloc(&ioFrame8[i], width * height * sizeof(unsigned char))); 
	}
		
	if (ioFrame32 == NULL)
		checkCudaErrors(hipMalloc(&ioFrame32, 4 * width * height * sizeof(unsigned char))); 

	if (devDx == NULL)
		checkCudaErrors(hipMalloc(&devDx, width * height * sizeof(float))); 

	if (devDy == NULL)
		checkCudaErrors(hipMalloc(&devDy, width * height * sizeof(float))); 

	if (devDt == NULL)
		checkCudaErrors(hipMalloc(&devDt, width * height * sizeof(float))); 

	if (cpuMem == NULL)
		cpuMem = (unsigned char*) malloc(3 * width * height * sizeof(unsigned char));
}

void kanadeNextFrame(unsigned char* pixels, unsigned width, unsigned height)
{
	for (unsigned i=0; i<PYRAMID_SIZE; ++i)
	{
		if (ioFrame8[i] != NULL)
		{
			if (prevFrame8[i] != NULL)
				hipFree(prevFrame8[i]);

			prevFrame8[i] = ioFrame8[i];
			ioFrame8[i] = NULL;
		}
	}

	allocateMemoryIfNeeded(width, height);

	if (frameTexMem != NULL) 
	{
		checkCudaErrors(hipUnbindTexture(frameTex));
		hipFreeArray(frameTexMem);
	}

	for (unsigned i=0; i<PYRAMID_SIZE; i++)
	{
		if (tex8bMem[i] != NULL)
		{
			checkCudaErrors(hipUnbindTexture(tex8b[i]));
			hipFreeArray(tex8bMem[i]);
		}
	}

	// przygotuj dane wejsciowe pod tekstury
	checkCudaErrors(hipMemcpy(ioFrame24, pixels, 3 * width * height * sizeof(unsigned char), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemset(ioFrame32, 0, 4 * width * height * sizeof(unsigned char)));
	prepareFrame<<<(width*height + BLOCK_DIM-1)/BLOCK_DIM, BLOCK_DIM>>>(ioFrame24, ioFrame32, ioFrame8[0], width, height);
	checkCudaErrors(hipGetLastError());

	// przygotuj tekstury
	hipChannelFormatDesc desc24b = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned); 	 
	checkCudaErrors(hipMallocArray(&frameTexMem, &desc24b, width, height)); 
	checkCudaErrors(hipDeviceSynchronize());	
	checkCudaErrors(hipMemcpyToArray(frameTexMem, 0, 0, ioFrame32, 4 * width * height * sizeof(unsigned char), hipMemcpyDeviceToDevice)); 
	checkCudaErrors(hipBindTextureToArray(frameTex, frameTexMem, desc24b)); 

	// @todo: przygotowywanie tekstur dla poszczegolnych leveli piramid, a nie tylko zerowego
	/*
	hipChannelFormatDesc desc8b = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned); 	 
	checkCudaErrors(hipMallocArray(&tex8bMem[0], &desc8b, width, height)); 
	checkCudaErrors(hipMemcpyToArray(tex8bMem[0], 0, 0, ioFrame8[0], width * height * sizeof(unsigned char), hipMemcpyDeviceToDevice)); 
	checkCudaErrors(hipBindTextureToArray(tex8b[0], tex8bMem[0], desc8b)); 
	*/
}

unsigned char* kanadeTranslate(unsigned width, unsigned height)
{	
	dim3 dimGrid((width + BLOCK_DIM - 1) / BLOCK_DIM, (height + BLOCK_DIM - 1) / BLOCK_DIM);
	dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);	

	hipMemset(ioFrame24, 0, 3*width*height*sizeof(unsigned char));
	translate<<<dimGrid, dimBlock>>>(20.5, 30.7, ioFrame24, width, height);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipMemcpy(cpuMem, ioFrame24, 3 * width * height * sizeof(unsigned char), hipMemcpyDeviceToHost));

	return cpuMem;
}

void kanadeCalculateG(unsigned pyrLvl, unsigned width, unsigned height, float& dxdx, float& dxdy, float& dydy)
{
	dim3 dimGrid((width + BLOCK_DIM - 1) / BLOCK_DIM, (height + BLOCK_DIM - 1) / BLOCK_DIM);
	dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);	
	calculate_dxdy<<<dimGrid, dimBlock>>>(ioFrame8[pyrLvl], devDx, devDy, width, height);
	checkCudaErrors(hipDeviceSynchronize());

	// oblicz dxdx, dxdy, dydy
}

void kanadeCalculateB(unsigned pyrLvl, float vx, float vy, unsigned width, unsigned height, float& b)
{
	dim3 dimGrid((width + BLOCK_DIM - 1) / BLOCK_DIM, (height + BLOCK_DIM - 1) / BLOCK_DIM);
	dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);	
	calculate_dt<<<dimGrid, dimBlock>>>(pyrLvl, vx, vy, prevFrame8[pyrLvl], devDt, width, height);
	checkCudaErrors(hipDeviceSynchronize());

	// oblicz b
}

// level 0 is automatically initiated elsewhere
void kanadeBuildPyramidLevel(unsigned levelId, unsigned newWidth, unsigned newHeight)
{
	dim3 dimGrid((newWidth + BLOCK_DIM - 1) / BLOCK_DIM, (newHeight + BLOCK_DIM - 1) / BLOCK_DIM);	
	dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);	

	build_pyramid_level<<<dimGrid, dimBlock>>>(ioFrame8[levelId-1], ioFrame8[levelId], newWidth, newHeight);

	// musimy sie synchronizowac, bo kolejne poziomy wymagaja obliczen z popzednich
	checkCudaErrors(hipDeviceSynchronize());
}

void kanadeInit()
{
	hipDeviceProp_t deviceProp;
	int devID = 0;
	checkCudaErrors(hipSetDevice(devID));
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);	

	// jesli tekstura odwoluje sie poza swoj zakres to zwracany jest kolor zerowy
	frameTex.addressMode[0] = hipAddressModeBorder;	
    frameTex.addressMode[1] = hipAddressModeBorder; 
}

void kanadeCleanup()
{
	if (frameTexMem != NULL)
		hipFreeArray(frameTexMem);

	for (unsigned i=0; i<PYRAMID_SIZE; i++)
	{
		if (ioFrame8[i] != NULL)
			hipFree(ioFrame8[i]);
		if (prevFrame8[i] != NULL)
			hipFree(prevFrame8[i]);
		if (tex8bMem[i] != NULL)
			hipFreeArray(tex8bMem[i]);
	}

	if (ioFrame24 != NULL)
		hipFree(ioFrame24);

	if (ioFrame32 != NULL)
		hipFree(ioFrame32);

	if (devDx != NULL)
		hipFree(devDx);

	if (devDy != NULL)
		hipFree(devDy);

	if (devDt != NULL)
		hipFree(devDt);

	if (cpuMem != NULL)
		free(cpuMem);
}

// for testing purposes only
void getIoFrame8(unsigned char* target, unsigned lvl, unsigned width, unsigned height) 
{
	checkCudaErrors(hipMemcpy(target, ioFrame8[lvl], width * height * sizeof(unsigned char), hipMemcpyDeviceToHost));
}

}

#endif